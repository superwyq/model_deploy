
#include <hip/hip_runtime.h>
#include<iostream>
// Kernel definition
__global__ void VecAdd(float* A, float* B, float* C)
{
    int i = threadIdx.x;
    C[i] = A[i] + B[i];
    printf("%f \n",C[i]);
}

int main()
{
    int dev = 0;
    hipSetDevice(dev);

    int N = 5;
    float A[N], B[N], C[N];
    // Initialize input vectors
    for(int i = 0; i < N; ++i)
    {
        A[i] = 1.0f;
        B[i] = 2.0f;
    }
    // Allocate vectors in device memory
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, N*sizeof(float));
    hipMalloc(&d_B, N*sizeof(float));
    hipMalloc(&d_C, N*sizeof(float));
    // Copy vectors from host memory to device memory
    hipMemcpy(d_A, A, N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, N*sizeof(float), hipMemcpyHostToDevice);

    // Kernel invocation with N threads
    VecAdd<<<1, N>>>(d_A, d_B, d_C);
    hipDeviceSynchronize();
    
    // Copy result from device memory to host memory
    // h_C contains the result in host memory
    hipMemcpy(C, d_C, N*sizeof(float), hipMemcpyDeviceToHost);
    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    // Free host memory

    return 0;
}
