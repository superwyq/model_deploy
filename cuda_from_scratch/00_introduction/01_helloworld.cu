
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void hello_from_gpu()
{
    printf("hello world from gpu\n");
}

int main(void)
{
    hello_from_gpu<<<4,4>>>();
    hipDeviceSynchronize();

    return 0;
}
