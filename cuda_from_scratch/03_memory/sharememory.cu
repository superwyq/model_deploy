#include <hip/hip_runtime.h>
#include <iostream>

#define BLOCK_WIDTH 16

__global__ void matrixMul(int *a, int *b, int *c, int width) {
  __shared__ int subTileA[BLOCK_WIDTH][BLOCK_WIDTH]; //share memory是可以声明在核函数内的
  __shared__ int subTileB[BLOCK_WIDTH][BLOCK_WIDTH];

  int row = blockIdx.y * BLOCK_WIDTH + threadIdx.y;
  int col = blockIdx.x * BLOCK_WIDTH + threadIdx.x;

  int sum = 0;
  
  for (int t = 0; t < width / BLOCK_WIDTH; ++t) { //初始化
    subTileA[threadIdx.y][threadIdx.x] =
        a[row * width + t * BLOCK_WIDTH + threadIdx.x];
    subTileB[threadIdx.y][threadIdx.x] =
        b[(t * BLOCK_WIDTH + threadIdx.y) * width + col];
    __syncthreads();

    for (int k = 0; k < BLOCK_WIDTH; ++k) {
      sum += subTileA[threadIdx.y][k] * subTileB[k][threadIdx.x];
    }
    __syncthreads();
  }

  c[row * width + col] = sum;
}

__global__ void matrixMul_worse(int *a, int *b, int *c, int width) {
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  int sum = 0;
  for (int i = 0; i < width; ++i) {
    sum += a[row * width + i] * b[i * width + col];
  }

  c[row * width + col] = sum;
}

int main() {
    const int width = 128;
    const int size = width * width * sizeof(int);

    int *h_a, *h_b, *h_c, *h_c2;
    h_a = new int[width * width];
    h_b = new int[width * width];
    h_c = new int[width * width];
    h_c2 = new int[width * width];

    // 初始化 h_a 和 h_b
    for (int i = 0; i < width; ++i) {
        h_a[i*width] = 2;
        h_b[i*width] = 2;
    }

    int *d_a, *d_b, *d_c, *d_c2;
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);
    hipMalloc((void **)&d_c2, size);

    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

    dim3 dimGrid(width / BLOCK_WIDTH, width / BLOCK_WIDTH);
    dim3 dimBlock(BLOCK_WIDTH, BLOCK_WIDTH);

    float runtime[2];
    hipEvent_t start, stop; //定义事件
    hipEventCreate(&start); //创建事件
    hipEventCreate(&stop);
    hipEventRecord(start); //记录一个开始事件
    hipEventSynchronize(start); //同步事件
    matrixMul<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, width);
    hipDeviceSynchronize();

    hipEventRecord(stop); //记录一个结束事件
    hipEventSynchronize(stop); //同步事件

    hipEventElapsedTime(&runtime[0], start, stop); //计算时间差
    std::cout << "Elapsed time 1: " << runtime[0] << " ms" << std::endl;
    hipEventDestroy(start); //销毁事件
    hipEventDestroy(stop);

    hipEventCreate(&start); //创建事件
    hipEventCreate(&stop);
    hipEventRecord(start); //记录一个开始事件
    hipEventSynchronize(start); //同步事件
    matrixMul_worse<<<dimGrid, dimBlock>>>(d_a, d_b, d_c2, width);
    hipDeviceSynchronize();
    hipEventRecord(stop); //记录一个结束事件
    hipEventSynchronize(stop); //同步事件
    hipEventElapsedTime(&runtime[1], start, stop); //计算时间差
    std::cout << "Elapsed time 2: " << runtime[1] << " ms" << std::endl;

    hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);
    hipMemcpy(h_c2, d_c2, size, hipMemcpyDeviceToHost);

    for (int i = 1; i < width; ++i) {
      std::cout << h_c[i*width]<< " ";
      if (i % 16 == 0) {
        std::cout << std::endl;
      }
    }

    for (int i = 0; i < width; ++i) {
      if (h_c[i] != h_c2[i]) {
        std::cout << "Error" << std::endl;
        break;
      }
    }

    // 处理结果

    delete[] h_a;
    delete[] h_b;
    delete[] h_c;
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
