#include <hip/hip_runtime.h>


#include <iostream>

void __global__ add(const double *x, const double *y, double *z, int N) {
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        z[i] = x[i] + y[i];
    }
}

int main(){

    const int N = 100000000;
    const int M = sizeof(double) * N;
    std::cout << "M: " << M << std::endl;
    double *h_x = (double *)malloc(M);
    double *h_y = (double *)malloc(M);
    double *h_z = (double *)malloc(M);

    double a = 1.1;
    double b = 2.2;
    for(int i = 0; i < N; i++){
        h_x[i] = a;
        h_y[i] = b;
    }

    double *d_x, *d_y, *d_z;
    hipMalloc((void **)&d_x, M);
    hipMalloc((void **)&d_y, M);
    hipMalloc((void **)&d_z, M);
    
    hipMemcpy(d_x, h_x, M, hipMemcpyHostToDevice);
    hipMemcpy(d_y, h_y, M, hipMemcpyHostToDevice);

    const int block_size = 256;
    const int grid_size = (N + block_size - 1) / block_size;

    hipEvent_t start, stop; //定义事件
    float elapsedTime[10]; //时间差
    for(int i = 0; i < 10; i++){
        hipEventCreate(&start); //创建事件
        hipEventCreate(&stop);
        hipEventRecord(start); //记录一个开始事件
        hipEventSynchronize(start); //同步事件

        add<<<grid_size, block_size>>>(d_x, d_y, d_z, N); //要计时的代码
        hipDeviceSynchronize();

        hipEventRecord(stop); //记录一个结束事件
        hipEventSynchronize(stop); //同步事件

        hipEventElapsedTime(&elapsedTime[i], start, stop); //计算时间差
        std::cout << "Elapsed time: " << elapsedTime[i] << " ms" << std::endl;
        hipEventDestroy(start); //销毁事件
        hipEventDestroy(stop);
    }
    hipMemcpy(h_z, d_z, M, hipMemcpyDeviceToHost);
    float average = 0;
    for(int i =0; i< 10; i++){
        average += elapsedTime[i];
    }
    average /= 10.0;
    std::cout << "Average time: " << average << " ms" << std::endl;
    return 0;
}