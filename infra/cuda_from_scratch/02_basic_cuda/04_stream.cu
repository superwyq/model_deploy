#include "hip/hip_runtime.h"
#include <iostream>

#define SPLIT 4

__global__ void vector_add(float *a, float *b, float *c, int width)
{
    if(threadIdx.x >= width) return;
    for(int i = 0; i < width; i++)
    {
        c[i] = a[i] + b[i];
    }
}

int main()
{
    int width = 1 << 15;
    float *a = new float[width];
    float *b = new float[width];
    float *c = new float[width];

    float *c_1 = c;
    float *c_2 = c + width / SPLIT;
    float *c_3 = c + 2 * width / SPLIT;

    float *d_a, *d_b, *d_c, *d_c_1, *d_c_2, *d_c_3;
    hipMalloc((void **)&d_a, width * sizeof(float));
    hipMalloc((void **)&d_b, width * sizeof(float));
    hipMalloc((void **)&d_c, width * sizeof(float));
    hipMalloc((void **)&d_c_1, width / SPLIT * sizeof(float));
    hipMalloc((void **)&d_c_2, width / SPLIT * sizeof(float));
    hipMalloc((void **)&d_c_3, width / SPLIT * sizeof(float));

    for(int i = 0;i<width; ++i){
        a[i] = 1.0f;
        b[i] = 2.0f;
    }
    hipMemcpy(d_a, a, width * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, width * sizeof(float), hipMemcpyHostToDevice);

    dim3 dimGrid(1, 1);
    dim3 dimBlock(width, 1);

    hipEvent_t start, stop;
    hipStream_t stream1, stream2, stream3;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);
    hipStreamCreate(&stream3);

    hipEventCreate(&start);
    hipEventRecord(start, 0);
    hipEventSynchronize(start);
    vector_add<<<dimGrid, dimBlock, 0, stream1>>>(d_a, d_b, d_c_1, width / SPLIT);
    vector_add<<<dimGrid, dimBlock, 0, stream2>>>(d_a + width / SPLIT, d_b + width / SPLIT, d_c_2, width / SPLIT);
    vector_add<<<dimGrid, dimBlock, 0, stream3>>>(d_a + 2 * width / SPLIT, d_b + 2 * width / SPLIT, d_c_3, width / SPLIT);
    hipStreamSynchronize(stream1);
    hipStreamSynchronize(stream2);
    hipStreamSynchronize(stream3);
    hipEventCreate(&stop);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    std::cout << "Elapsed time: " << elapsedTime << " ms" << std::endl;

    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipEventCreate(&start);
    hipEventRecord(start, 0);
    hipEventSynchronize(start);
    vector_add<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, width);
    hipEventCreate(&stop);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    std::cout << "Elapsed time: " << elapsedTime << " ms" << std::endl;

    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);
    hipStreamDestroy(stream3);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c_1);
    hipFree(d_c_2);
    hipFree(d_c_3);

    delete[] a;
    delete[] b;
    delete[] c;

    return 0;

}