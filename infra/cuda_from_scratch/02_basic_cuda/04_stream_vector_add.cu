
#include <hip/hip_runtime.h>
#include <iostream>

#define SPLIT 4

__global__ void vector_add(float *a, float *b, float *c, int width){
    int index = (blockIdx.x * gridDim.y + blockIdx.y)*blockDim.x * blockDim.y + threadIdx.x * blockDim.y + threadIdx.y;
    // 计算线程的全局索引，第blockIdx.x行第blockIdx.y列的block中的第threadIdx.x行第threadIdx.y列的线程
    if (index >= width) return;
    c[index] = a[index] + b[index];
    
}

void single_stream(float *a, float *b, float *c, int width){
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start,0); //0是指默认流
    hipEventSynchronize(start);

    float *d_a, *d_b, *d_c;
    hipMalloc((void **)&d_a, width * sizeof(float));
    hipMalloc((void **)&d_b, width * sizeof(float));
    hipMalloc((void **)&d_c, width * sizeof(float));

    hipMemcpy(d_a, a, width * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, width * sizeof(float), hipMemcpyHostToDevice);

    dim3 dimGrid(1, width >> 10);
    dim3 dimBlock(256, 4);
    vector_add<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, width);
    hipMemcpy(c, d_c, width * sizeof(float), hipMemcpyDeviceToHost);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float time;
    hipEventElapsedTime(&time, start, stop);
    std::cout << "Single stream time: " << time << "ms" << std::endl;

    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}

void multi_stream(float *a, float *b, float *c, int width){
    hipStream_t stream1, stream2, stream3,stream4;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);
    hipStreamCreate(&stream3);
    hipStreamCreate(&stream4);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    hipEventSynchronize(start);

    float *d_a, *d_b, *d_c_1, *d_c_2, *d_c_3, *d_c_4;

    hipMalloc((void **)&d_a, width * sizeof(float));
    hipMalloc((void **)&d_b, width * sizeof(float));
    hipMalloc((void **)&d_c_1, width / SPLIT * sizeof(float));
    hipMalloc((void **)&d_c_2, width / SPLIT * sizeof(float));
    hipMalloc((void **)&d_c_3, width / SPLIT * sizeof(float));
    hipMalloc((void **)&d_c_4, width / SPLIT * sizeof(float));

    dim3 dimGrid(1, 1);
    dim3 dimBlock(width / SPLIT, 1);
    
    hipMemcpyAsync(d_a, a, width / SPLIT * sizeof(float), hipMemcpyHostToDevice, stream1);
    hipMemcpyAsync(d_b, b, width / SPLIT * sizeof(float), hipMemcpyHostToDevice, stream1);
    vector_add<<<dimGrid, dimBlock, 0, stream1>>>(d_a, d_b, d_c_1, width / SPLIT);
    hipMemcpyAsync(c, d_c_1, width / SPLIT * sizeof(float), hipMemcpyDeviceToHost, stream1);

    hipMemcpyAsync(d_a + width / SPLIT, a + width / SPLIT, width / SPLIT * sizeof(float), hipMemcpyHostToDevice, stream2);
    hipMemcpyAsync(d_b + width / SPLIT, b + width / SPLIT, width / SPLIT * sizeof(float), hipMemcpyHostToDevice, stream2);
    vector_add<<<dimGrid, dimBlock, 0, stream2>>>(d_a + width / SPLIT, d_b + width / SPLIT, d_c_2, width / SPLIT);
    hipMemcpyAsync(c + width / SPLIT, d_c_2, width / SPLIT * sizeof(float), hipMemcpyDeviceToHost, stream2);

    hipMemcpyAsync(d_a + 2 * width / SPLIT, a + 2 * width / SPLIT, width / SPLIT * sizeof(float), hipMemcpyHostToDevice, stream3);
    hipMemcpyAsync(d_b + 2 * width / SPLIT, b + 2 * width / SPLIT, width / SPLIT * sizeof(float), hipMemcpyHostToDevice, stream3);
    vector_add<<<dimGrid, dimBlock, 0, stream3>>>(d_a + 2 * width / SPLIT, d_b + 2 * width / SPLIT, d_c_3, width / SPLIT);
    hipMemcpyAsync(c + 2 * width / SPLIT, d_c_3, width / SPLIT * sizeof(float), hipMemcpyDeviceToHost, stream3);

    hipMemcpyAsync(d_a + 3 * width / SPLIT, a + 3 * width / SPLIT, width / SPLIT * sizeof(float), hipMemcpyHostToDevice, stream4);
    hipMemcpyAsync(d_b + 3 * width / SPLIT, b + 3 * width / SPLIT, width / SPLIT * sizeof(float), hipMemcpyHostToDevice, stream4);
    vector_add<<<dimGrid, dimBlock, 0, stream4>>>(d_a + 3 * width / SPLIT, d_b + 3 * width / SPLIT, d_c_4, width / SPLIT);
    hipMemcpyAsync(c + 3 * width / SPLIT, d_c_4, width / SPLIT * sizeof(float), hipMemcpyDeviceToHost, stream4);

    hipStreamSynchronize(stream1);
    hipStreamSynchronize(stream2);
    hipStreamSynchronize(stream3);
    hipStreamSynchronize(stream4);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float time;
    hipEventElapsedTime(&time, start, stop);
    std::cout << "Multi stream time: " << time << "ms" << std::endl;

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c_1);
    hipFree(d_c_2);
    hipFree(d_c_3);
    hipFree(d_c_4);
}

bool if_right(float *c, int width){
    for(int i = 0; i < width; i++){
        if(c[i] != 3.0f){
            std::cout << "Error at: " << i << std::endl;
            return false;
        }
    }
    return true;
}

int main(){
    int width = 1 << 12;
    float *a = new float[width];
    float *b = new float[width];
    float *c = new float[width];

    for(int i = 0; i < width; i++){
        a[i] = 1.0f;
        b[i] = 2.0f;
    }

    single_stream(a, b, c, width);
    if_right(c, width);
    multi_stream(a, b, c, width);
    if_right(c, width);

    return 0;
}